#include "cnn.cuh"
#include "matrix.h"
#include "cuda_matrix.cuh"
#include "util.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

void initConvolutionalModel(ConvolutionalModel** model, int batchSize, float learningRate) {  
    *model = (ConvolutionalModel*) calloc(1, sizeof(ConvolutionalModel));
    if (!model) { perror("calloc model"); exit(1); }

    ConvolutionalNetwork *cnn = (ConvolutionalNetwork*) malloc(sizeof(ConvolutionalNetwork));
    if (!cnn) { perror("malloc network"); exit(1); }
    cnn->numLayers = 0;
  
    model->network = cnn;
    model->learningRate = learningRate;
    model->batchSize = batchSize;
  
    checkError("Init CNN");
  }

ConvolutionalLayer* createConvolutionalLayer(int batch_size, int outChannels,
        int outputRows, int outputCols, ConvolutionalLayer* prev, hiprandState_t* state) {
    ConvolutionalLayer* layer = (ConvolutionalLayer*) calloc(1, sizeof(ConvolutionalLayer));
    if (!layer) {perror("malloc"); exit(1);}
    layer->outChannels = outChannels;
    layer->outputRows = outputRows;
    layer->outputCols = outputCols;
    layer->prev = prev;
    if (prev != NULL) {
        prev->next = layer;
        layer->inChannels = prev->outChannels;
        layer->imgRows = prev->outputRows;
        layer->imgCols = prev->outputCols;
    }

    // TODO: this is assuming stride = 0
    layer->kernelRows = layer->imgRows + 1 - outputRows;
    layer->kernelCols = layer->imgCols + 1 - outputCols;

    initRandomTensor4D(&layer->filters, outChannels, layer->inChannels,
        layer->kernelRows, layer->kernelCols, state);
    initRandomVector(&layer->biases, outChannels, state);
    initTensor4D(&layer->outputs, batch_size, outChannels, outputRows, outputCols);

    return layer;
}

void addInputLayer(ConvolutionalModel *model, int channels, int rows, int cols, hiprandState_t* state) {
    model->inChannels = channels;
    model->inHeight = rows;
    model->inWidth = cols;
    model->outChannels = channels;
    model->outHeight = rows;
    model->outWidth = cols;
    ConvolutionalLayer* layer = createConvolutionalLayer(model->batchSize, channels,
        rows, cols, NULL, state);
    model->network->input = layer;
    model->network->layers = layer;
    model->network->output = layer;
}

void addConvLayer(ConvolutionalModel *model, int channels, int rows, int cols, hiprandState_t* state) {
    ConvolutionalLayer* prev = model->network->output;
    ConvolutionalLayer* layer = createConvolutionalLayer(model->batchSize, channels,
        rows, cols, prev, state);
    model->network->numLayers++;
    model->network->output = layer;
    model->outChannels = channels;
    model->outHeight = rows;
    model->outWidth = cols;
}

// void layerForward(ConvolutionalLayer *layer, int sampleNo) {
//     /* for each channel of this input sample, do forward pass */

//     // TODO: change size when we have image and kernel dimensions
//     int imgRows = layer->imgRows;
//     int imgCols = layer->imgCols;
//     int imgSize = imgRows * imgCols;
//     int kernelRows = layer->kernelRows;
//     int kernelCols = layer->kernelCols;
    
//     int output_channels = layer->outChannels;
//     int input_channels = layer->inChannels;

//     int c, outChannels;
//     Matrix *temp;
//     initMatrix(&temp, layer->outputRows, layer->outputCols);

//     Matrix **inputImages, *outputImageK, **filtersK;
//     inputImages = (layer->prev->outputs)[sampleNo];

//     for (outChannels = 0; k < output_channels; k++) {
//         outputImageK = (layer->outputs)[sampleNo][outChannels];
//         filtersK = (layer->filters)[outChannels];

//         // TODO: change to convolution
//         /* convolve first input channel image with first filter */
//         deviceConvolve(inputImages[0], imgRows, imgCols, 
//             filtersK[0], kernelRows, kernelCols,
//             outputImageK, 1, 0);
//         for (c = 1; c < input_channels; c++) {
//             /* for each remaining channel, add the convolution of the image 
//              * and filter to the running total
//              */
//             deviceConvolve(inputImages[c], imgRows, imgCols,
//                 filtersK[c], kernelRows, kernelCols,
//                 temp, 1, 0);

//             deviceMatrixAdd(
//                 outputImageK,
//                 temp,
//                 outputImageK,
//                 imgSize
//             );
//         }
        
//         /* add bias to every element */
//         deviceMatrixAddScalarElementwise(outputImageK, outputImageK, (layer->biases)[outChannels], imgSize);

//         /* apply sigmoid activation to every element */
//         deviceSigmoid(outputImageK, outputImageK, imgSize);
//     }

//     freeMatrix(temp);
//   }
  
// /**
//  * :param input: list of input samples (size of minibatch).
//  * Each input sample is an image with inChannels channels, or a list of inChannels lists of floats.
//  */
// void forward(ConvolutionalModel *model, float ***input) {
//     ConvolutionalNetwork net = *(model->network);
//     int batchSize = model->batchSize;
//     int inputChannels = model->inChannels;
//     int imageSize = model->inHeight * model->inWidth;

//     int i, j;

//     /* initialize 4D tensor of input images */
//     for (i = 0; i < batchSize; ++i) {
//         for (j = 0; j < inputChannels; ++j) {
//             setDeviceMatrixData((net.layers->outputs)[i][j], input[i][j], imageSize);
//         }
//     }
    
//     ConvolutionalLayer *curr = net.layers->next; /* first hidden layer */
//     for (i = 0; i < net.numLayers; ++i) {
//         if (!curr) break;

//         /* for each sample in minibatch, go forward */
//         for (j = 0; j < batchSize; ++j) {
//             layerForward(curr, j);
//         }
//         curr = curr->next;
//     }
//   }


// void initLayerGradients(ConvolutionalLayer *layer, int batchSize) {
//     int i, j;
//     int outChannels = layer->outChannels;
//     int r = layer->outputRows;
//     int c = layer->outputCols;

//     /* backprop fields needs batchsize arrays of outChannels arrays of pointers to Matrix on the device */
//     layer->gradient = (Matrix***) malloc(sizeof(Matrix**) * batchSize);
//     if (!(layer->gradient)) {perror("malloc layer g"); exit(1);}

//     layer->delta = (Matrix***) malloc(sizeof(Matrix**) * batchSize);
//     if (!(layer->delta)) {perror("malloc layer d"); exit(1);}
//     if (layer->prev) {
//         layer->error = (Matrix***) malloc(sizeof(Matrix**) * batchSize);
//         if (!(layer->error)) {perror("malloc layer e"); exit(1);}
//     }
    
//     for (i = 0; i < batchSize; i++) {
//         layer->gradient[i] = (Matrix**) malloc(sizeof(Matrix*) * outChannels);
//         if (!(layer->gradient[i])) {perror("malloc layer g"); exit(1);}

//         layer->delta[i] = (Matrix**) malloc(sizeof(Matrix*) * outChannels);
//         if (!(layer->delta[i])) {perror("malloc layer d"); exit(1);}

//         layer->error[i] = (Matrix**) malloc(sizeof(Matrix*) * outChannels);
//         if (!(layer->error[i])) {perror("malloc layer e"); exit(1);}
        
//         for (j = 0; j < outChannels; j++) {
//             initMatrix(layer->gradient[i] + j, r, c);
//             initMatrix(layer->delta[i] + j, r, c);
//             initMatrix(layer->error[i] + j, r, c);
//         }
//     }
// }

// void compileModel(ConvolutionalModel *model) {
//     ConvolutionalLayer* curr = model->network->layers->next;
//     for (int i = 0; i < model->network->numLayers; ++i) {
//         if (!curr) break;
//         initLayerGradients(curr, model->batchSize);
//         curr = curr->next;
//     }
// }

// void layerBackward(ConvolutionalLayer* layer, ConvolutionalModel* model) {
//     int batchSize = model->batchSize;
//     int outChannels = layer->outChannels;
//     int r = layer->outputRows;
//     int c = layer->outputCols;
//     int outputSize = r * c;
//     int i, j;

//     for (i = 0; i < batchSize; ++i) {
//         for (j = 0; j < outChannels; ++j) {
//             deviceSigmoidOutputDerivative(layer->outputs[i][j], layer->gradient[i][j], outputSize);
//             deviceHadamardProd(layer->gradient[i][j], layer->error[i][j], layer->gradient[i][j], outputSize);
//         }
//     }
    
// }

// void layerUpdate(ConvolutionalLayer* layer, int batchSize) {
//     return;
// }

// void backward(ConvolutionalModel* model, float*** targets) {
//     ConvolutionalNetwork* net = model->network;
//     int batchSize = model->batchSize;
//     ConvolutionalLayer* curr = net->output;
//     int outputSize = curr->outputRows * curr->outputCols;
//     int i, j;
//     for (i = 0; i < batchSize; ++i) {
//         for (j = 0; j < curr->outChannels; ++j) {
//             setDeviceMatrixData(curr->error[i][j], targets[i][j], outputSize);
//             deviceMatrixSub(curr->outputs[i][j], curr->error[i][j], curr->error[i][j], outputSize);
//             deviceMatrixDivideScalarElementwise(curr->error[i][j], curr->error[i][j], outputSize, outputSize);
//         } 
//     }

//     for (int i = 0; i < net->numLayers; ++i) {
//         if (!curr->prev) break;
//         layerBackward(curr, model);
//         curr = curr->prev;
//       }
//       curr = net->output;
//       for (int i = 0; i < net->numLayers; ++i) {
//         if (!curr->prev) break;
//         layerUpdate(curr, batchSize);
//         curr = curr->prev;
//       }
// }