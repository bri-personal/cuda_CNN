#include "cuda_cnn.cuh"
#include "matrix.h"
#include "cuda_matrix.cuh"
#include "util.h"
#include "cnn.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


ConvolutionalLayer* createConvolutionalLayer(int batch_size, int outChannels,
        int outputRows, int outputCols, ConvolutionalLayer* prev, hiprandState_t* state) {
    ConvolutionalLayer* layer = (ConvolutionalLayer*) calloc(1, sizeof(ConvolutionalLayer));
    if (!layer) {perror("malloc"); exit(1);}
    layer->outChannels = outChannels;
    layer->outputRows = outputRows;
    layer->outputCols = outputCols;
    layer->prev = prev;
    if (prev != NULL) {
        prev->next = layer;
        layer->inChannels = prev->outChannels;
        layer->imgRows = prev->outputRows;
        layer->imgCols = prev->outputCols;
    }

    // TODO: this is assuming stride = 1 and padding = 0
    layer->kernelRows = layer->imgRows + 1 - outputRows;
    layer->kernelCols = layer->imgCols + 1 - outputCols;

    if(layer->inChannels > 0) {
        initRandomTensor4D(&layer->filters, outChannels, layer->inChannels,
            layer->kernelRows, layer->kernelCols, state);
    }
    initRandomVector(&layer->biases, outChannels, state);
    initTensor4D(&layer->outputs, batch_size, outChannels, outputRows, outputCols);

    return layer;
}

void addInputLayer(ConvolutionalModel *model, int channels, int rows, int cols, hiprandState_t* state) {
    model->inChannels = channels;
    model->inHeight = rows;
    model->inWidth = cols;
    model->outChannels = channels;
    model->outHeight = rows;
    model->outWidth = cols;
    ConvolutionalLayer* layer = createConvolutionalLayer(model->batchSize, channels,
        rows, cols, NULL, state);
    model->network->input = layer;
    model->network->layers = layer;
    model->network->output = layer;
}

void addConvLayer(ConvolutionalModel *model, int channels, int rows, int cols, hiprandState_t* state) {
    ConvolutionalLayer* prev = model->network->output;
    ConvolutionalLayer* layer = createConvolutionalLayer(model->batchSize, channels,
        rows, cols, prev, state);
    model->network->numLayers++;
    model->network->output = layer;
    model->outChannels = channels;
    model->outHeight = rows;
    model->outWidth = cols;
}

void layerForward(ConvolutionalLayer *layer, int batchSize) {
    /* for each channel of this input sample, do forward pass */
    int outChannels = layer->outChannels;
    int outRows = layer->outputRows;
    int outCols = layer->outputCols;

    int im2colOutRows = batchSize * outRows * outCols;
    int im2colOutArea = im2colOutRows * outChannels;

    Matrix* temp;
    initMatrix(&temp, im2colOutRows, outChannels);

    deviceConvolve(
        layer->prev->outputs,
        layer->filters,
        temp,
        0, 1, //TODO: actually use padding and stride
        layer->inChannels,
        layer->imgRows,
        layer->imgCols,
        outChannels,
        layer->kernelRows,
        layer->kernelCols,
        im2colOutRows,
        outChannels
    );

    /* add bias for each out channel to every element in that channel */
    deviceMatrixAddScalarColumnwise(temp, temp, layer->biases, im2colOutRows, outChannels);

    /* apply sigmoid activation to every element */
    deviceSigmoid(temp, temp, im2colOutArea);

    /* put temp contents into layer's output Tensor4D */
    deviceReorderIm2ColToConv(temp, layer->outputs, im2colOutArea);

    freeMatrix(temp);
  }
  
/**
 * :param input: list of input samples (size of minibatch).
 * Each input sample is an image with inChannels channels, or a list of inChannels lists of floats.
 */
void forward(ConvolutionalModel *model, Tensor4D* input) {
    ConvolutionalNetwork* net = model->network;
    int batchSize = model->batchSize;
    int inputChannels = model->inChannels;
    int imageSize = model->inHeight * model->inWidth;
    int inputSize = batchSize*inputChannels*imageSize;

    /* initialize 4D tensor of input images */
    setDeviceTensor4DData(net->input->outputs, input->data, inputSize);
    
    ConvolutionalLayer *curr = net->input->next; /* first hidden layer */
    while (curr != NULL) {
        layerForward(curr, batchSize);
        curr = curr->next;
    }
  }


void initLayerGradients(ConvolutionalLayer *layer, int batchSize) {
    int i, j;
    int outChannels = layer->outChannels;
    int rows = layer->outputRows;
    int cols = layer->outputCols;

    initTensor4D(layer->gradient, batchSize, outChannels, rows, cols);

    initTensor4D(layer->delta, batchSize, outChannels, rows, cols);

    if (layer->prev) {
        initTensor4D(layer->error, batchSize, outChannels, rows, cols);
    } else {
        layer->error = NULL;
    }
}

void compileModel(ConvolutionalModel *model) {
    int batchSize = model->batchSize;

    ConvolutionalLayer* curr = model->network->layers->next;
    while (curr != NULL) {
        initLayerGradients(curr, batchSize);
        curr = curr->next;
    }
}

// TODO: make for Tensor4D
// void layerBackward(ConvolutionalLayer* layer, ConvolutionalModel* model) {
//     int batchSize = model->batchSize;
//     int outChannels = layer->outChannels;
//     int r = layer->outputRows;
//     int c = layer->outputCols;
//     int outputSize = r * c;
//     int i, j;

//     for (i = 0; i < batchSize; ++i) {
//         for (j = 0; j < outChannels; ++j) {
//             deviceSigmoidOutputDerivative(layer->outputs[i][j], layer->gradient[i][j], outputSize);
//             deviceHadamardProd(layer->gradient[i][j], layer->error[i][j], layer->gradient[i][j], outputSize);
//         }
//     }
    
// }

void layerUpdate(ConvolutionalLayer* layer, int batchSize) {
    // TODO
    return;
}

// TODO: make for Tensor4D
// void backward(ConvolutionalModel* model, Tensor4D* targets) {
//     ConvolutionalNetwork* net = model->network;
//     int batchSize = model->batchSize;
//     ConvolutionalLayer* curr = net->output;
//     int outputSize = curr->outputRows * curr->outputCols;

//     int i, j;
//     for (i = 0; i < batchSize; ++i) {
//         for (j = 0; j < curr->outChannels; ++j) {
//             setDeviceMatrixData(curr->error[i][j], targets[i][j], outputSize);
//             deviceMatrixSub(curr->outputs[i][j], curr->error[i][j], curr->error[i][j], outputSize);
//             deviceMatrixDivideScalarElementwise(curr->error[i][j], curr->error[i][j], outputSize, outputSize);
//         } 
//     }

//     for (int i = 0; i < net->numLayers; ++i) {
//         if (!curr->prev) break;
//         layerBackward(curr, model);
//         curr = curr->prev;
//       }
//       curr = net->output;
//       for (int i = 0; i < net->numLayers; ++i) {
//         if (!curr->prev) break;
//         layerUpdate(curr, batchSize);
//         curr = curr->prev;
//       }
// }