#include "hip/hip_runtime.h"
#include "util.h"
#include "matrix.h"
#include <stdio.h>
#include <string.h>
#include <assert.h>

void printArray(float *a, int n) {
  printf("[ ");
  for (int i = 0; i < n; ++i)
    printf("%d ", (int)a[i]);
  printf("]\n");
}

void test_matrixMult() {
  // (4,2)*(2,3) = (4,3)
  float a[8] = {-2,-1,
                 0, 1,
                 2, 3,
                 4, 5};
  float b[6] = {-1,0,1,
                 0,1,0};
  Matrix *A, *B, *C;
  initMatrix(&A, 4, 2);
  initMatrix(&B, 2, 3);
  initMatrix(&C, 4, 3);
  setDeviceMatrixData(A, a, 8);
  setDeviceMatrixData(B, b, 6);

  deviceMatrixMult(A, B, C, 12);

  float c[12];
  getDeviceMatrixData(c, C, 12);

  char result[64];
  char expected[64] = "2 -1 -2 0 1 0 -2 3 2 -4 5 4";
  int offset = 0;
  for (int i = 0; i < 12; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)c[i]);
  }
  printf("Testing matrix mult\n");
  printf("Result: %s\n", result);
  printf("Expect: %s\n", expected);
  if (strncmp(result, expected, 27) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }
  printf("\nPASSED\n\n");

  freeMatrix(A);
  freeMatrix(B);
  freeMatrix(C);
}

void test_matrixElementWise() {
  Matrix *A, *B, *C;
  float data[6] = {0,1,2,3,4,5};

  initMatrix(&A, 2, 3);
  initMatrix(&B, 2, 3);
  initMatrix(&C, 2, 3);
  setDeviceMatrixData(A, data, 6);
  setDeviceMatrixData(B, data, 6);

  deviceMatrixAdd(A, B, C, 6);

  float c[6];
  getDeviceMatrixData(c, C, 6);

  char result[64];
  int offset = 0;
  for (int i = 0; i < 6; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)c[i]);
  }
  printf("Testing matrix add\n");
  printf("Result: %s\n", result);
  printf("Expect: 0 2 4 6 8 10\n");
  if (strncmp(result, "0 2 4 6 8 10", 12) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }

  deviceMatrixSub(A, B, C, 6);
  getDeviceMatrixData(c, C, 6);

  offset = 0;
  for (int i = 0; i < 6; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)c[i]);
  }
  printf("Testing matrix sub\n");
  printf("Result: %s\n", result);
  printf("Expect: 0 0 0 0 0 0\n");
  if (strncmp(result, "0 0 0 0 0 0", 11) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }

  deviceHadamardProd(A, B, C, 6);
  getDeviceMatrixData(c, C, 6);

  offset = 0;
  for (int i = 0; i < 6; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)c[i]);
  }
  printf("Testing hadamardProd \n");
  printf("Result: %s\n", result);
  printf("Expect: 0 1 4 9 16 25\n");
  if (strncmp(result, "0 1 4 9 16 25", 13) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }

  deviceSigmoid(C, C, 6);
  getDeviceMatrixData(c, C, 6);

  offset = 0;
  for (int i = 0; i < 6; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%.2f ", c[i]);
  }
  printf("Testing sigmoid \n");
  printf("Result: %s\n", result);
  printf("Expect: 0.50 0.73 0.98 1.00 1.00 1.00\n");
  if (strncmp(result, "0.50 0.73 0.98 1.00 1.00 1.00", 28) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }
  printf("\nPASSED\n\n");


  deviceMatrixScale(A, 2, C, 6);
  getDeviceMatrixData(c, C, 6);

  offset = 0;
  for (int i = 0; i < 6; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)c[i]);
  }
  printf("Testing scalar \n");
  printf("Result: %s\n", result);
  printf("Expect: 0 2 4 6 8 10\n");
  if (strncmp(result, "0 2 4 6 8 10", 12) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }
  printf("\nPASSED\n\n");

  freeMatrix(A);
  freeMatrix(B);
  freeMatrix(C);
}

void test_transpose() {
  Matrix *A, *tA, *C;
  float a[8] = {
    0,1,2,3,
    4,5,6,7
  };
  initMatrix(&A, 2, 4);     // A (2,4)
  setDeviceMatrixData(A, a, 8);
  matrixTranpose(A, &tA, 2, 4);   // tA (4,2)

  initMatrix(&C, 2, 2);
  deviceMatrixMult(A, tA, C, 4);  // (2,4)(4,2) = (2,2)

  float c[12];
  getDeviceMatrixData(c, C, 4);

  char result[32];
  char expected[32] = "14 38 38 126";
  int offset = 0;
  for (int i = 0; i < 4; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)c[i]);
  }
  printf("Testing matrix transpose\n");
  printf("Result: %s\n", result);
  printf("Expect: %s\n", expected);
  if (strncmp(result, expected, strlen(expected)) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }
  printf("\nPASSED\n\n");

  freeMatrix(A);
  freeMatrix(tA);
  freeMatrix(C);
}

void test_acrossRows() {
  Matrix *A, *B, *C;
  float a[6] = {1,2,3,
                4,5,6};
  float b[3] = {5,5,5};
  printf("Testing matrix add vec\n");
  initMatrix(&A, 2, 3);
  setDeviceMatrixData(A, a, 6);
  initMatrix(&B, 1, 3);
  setDeviceMatrixData(B, b, 3);
  initMatrix(&C, 2, 3);
  deviceMatrixAddVec(A, B, C, 6);
  float c[6];
  getDeviceMatrixData(c, C, 6);
  float exp[6] = {6,7,8,9,10,11};
  printArray(exp, 6);
  printArray(c, 6);
  for (int i = 0; i < 6; ++i)
    assert(c[i] == (a[i] + 5));
  printf("\nPASSED\n\n");
  printf("Testing matrix reduce rows\n");
  deviceMatrixReduceRows(A, B, 2, 3);
  getDeviceMatrixData(b, B, 3);
  float exp2[3] = {5,7,9};
  printArray(exp2, 3);
  printArray(b, 3);
  for (int i = 0; i < 3; ++i)
    assert(b[i] == (a[i] + a[i+3]));
  printf("\nPASSED\n\n");
}

void test_unfold() {
    Matrix *Img, *Kernel, *Unfolded;
    float img[9] = {
      0,1,2,
      3,4,5,
      6,7,8
    };
    initMatrix(&Img, 3, 3);
    setDeviceMatrixData(Img, img, 9);

    float kernel[4] = {
        0, 1,
        2, 3
    };
    initMatrix(&Kernel, 2, 2);
    setDeviceMatrixData(Kernel, kernel, 4);

    int resRows = 3 - 2 + 1;
    int resCols = 3 - 2 + 1;

    deviceUnfoldMatrix(Img, &Unfolded, 2, 2, resRows, resCols);
  
    float unfolded[16];
    getDeviceMatrixData(unfolded, Unfolded, 16);
  
    char result[64];
    char expected[64] = "0 1 3 4 1 2 4 5 3 4 6 7 4 5 7 8";
    int offset = 0;
    for (int i = 0; i < 16; ++i) {
      offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)unfolded[i]);
    }
    printf("Testing matrix unfold\n");
    printf("Result: %s\n", result);
    printf("Expect: %s\n", expected);
    if (strncmp(result, expected, strlen(expected)) != 0) {
      printf("FAILED\n");
      exit(EXIT_FAILURE);
    }

    printf("\nPASSED\n\n");
  
    freeMatrix(Img);
    freeMatrix(Kernel);
    freeMatrix(Unfolded);
  }

  void test_convolve() {
    Matrix *Img, *Kernel, *Convolved;
    float img[9] = {
      0,1,2,
      3,4,5,
      6,7,8
    };
    initMatrix(&Img, 3, 3);
    setDeviceMatrixData(Img, img, 9);

    float kernel[4] = {
        0, 1,
        1, 0
    };
    initMatrix(&Kernel, 2, 2);
    setDeviceMatrixData(Kernel, kernel, 4);

    deviceConvolve(Img, 3, 3, Kernel, 2, 2, &Convolved, 1, 0);
  
    float convolved[4];
    getDeviceMatrixData(convolved, Convolved, 4);
  
    char result[32];
    char expected[32] = "4 6 10 12";
    int offset = 0;
    for (int i = 0; i < 4; ++i) {
      offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)convolved[i]);
    }
    printf("Testing matrix unfold\n");
    printf("Result: %s\n", result);
    printf("Expect: %s\n", expected);
    if (strncmp(result, expected, strlen(expected)) != 0) {
      printf("FAILED\n");
      exit(EXIT_FAILURE);
    }

    printf("\nPASSED\n\n");
  
    freeMatrix(Img);
    freeMatrix(Kernel);
    freeMatrix(Convolved);
  }

int main() {

  test_matrixMult();
  test_matrixElementWise();
  test_transpose();
  test_acrossRows();
  test_unfold();
  test_convolve();

  return 0;
}